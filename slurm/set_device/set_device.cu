#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void hello()
{
        printf("hello world, Im thread %d on block %d\n", threadIdx.x, blockIdx.x);
}

int main(int argc, char *argv[]) {
    int deviceId, nDevices, taskID;
    hipError_t err;
    hipDeviceProp_t prop;

    if( argc == 3 ) {
        printf("The argument supplied are device-ID: %s and task-ID: %s\n",
               argv[1], argv[2]);
        deviceId = atoi(argv[1]);
        taskID = atoi(argv[2]);
    } else {
        printf("Usage: set_device.o device-id-integer n-taks-integer\n");
        return 1;
    }

    err = hipSetDevice(deviceId);
    if (err != hipSuccess) {
        printf("API error %s:%d Returned:%d\n", __FILE__, __LINE__, err);
        return 1;
    }

    printf("Task id: %d - Desired DeviceId: %d\n", taskID, deviceId);
    hipGetDeviceCount(&nDevices);
    printf("Number of devices available: %d\n", nDevices);
    hipGetDeviceProperties(&prop, deviceId);
    printf("Device Number: %d\n", deviceId);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("Finally...");
    int num_threads = 2;
    int num_blocks = 2;
    hello<<<num_blocks,num_threads>>>();
    hipDeviceSynchronize();
    return 0;
}
